#include "StereoMatch.cuh"
#include "kernels/StereoMatchKernels.cuh"
#include "kernels/ColorConverterKernels.cuh"
#include "errorCheck.cuh"

cCudaStereoMatcher::cCudaStereoMatcher() {
	isInitialized = false;
	grayscaleBufferUsed = false;
	imgSize = 0;
	width = 0;
	height = 0;
	kernelSize = 0;
	maxDisp = 0;
	consistencyTreshold = 0;
	modes = OP_INPUT_GRAYSCALE;

	host_grayLeft = NULL;
	host_grayRight = NULL;
	host_colorLeft = NULL;
	host_colorRight = NULL;
	host_dispColorLeft = NULL;
	host_dispColorRight = NULL;
	host_dispRawLeft = NULL;
	host_dispRawRight = NULL;

	dev_colorLeft = NULL;
	dev_colorRight = NULL;
	dev_grayLeft = NULL;
	dev_grayRight = NULL;
	dev_dispColorLeft = NULL;
	dev_dispColorRight = NULL;
	dev_dispRawLeft = NULL;
	dev_dispRawRight = NULL;
}
cCudaStereoMatcher::~cCudaStereoMatcher() {
	deinitSystem();
}
bool cCudaStereoMatcher::initSystem(int width, int height, int modes) {

	if(isInitialized)
		deinitSystem();

	printf("width: %d, height: %d, mode: %d\n",width,height,modes);

	if((modes & OP_INPUT_COLOR) && (modes & OP_INPUT_GRAYSCALE)){
		printf("Invalid operation mode. \n");
		return false;
	}

	if(width <= 0 || height <= 0 || modes < 1){
		printf("Invalid mode or size. \n");
		return false;
	}

	this->width = width;
	this->height = height;
	this->imgSize = width*height;
	this->modes = modes;

	blockSize = 1024;
	blockCnt = imgSize / blockSize + (imgSize % blockSize == 0 ? 0 : 1);
	printf("BlockCnt: %d, BlockSize: %d\n",blockCnt,blockSize);

	// Farbbild als Eingabe aber kein color match -> Grayscale erzeugen
	if((modes & OP_INPUT_COLOR) && !(modes & OP_COLOR_MATCH)){
		grayscaleBufferUsed = true;
	}
	else{
		grayscaleBufferUsed = false;
	}

	host_grayLeft = new unsigned char[imgSize];
	host_grayRight = new unsigned char[imgSize];
	host_colorLeft = new unsigned char[imgSize * 3];
	host_colorRight = new unsigned char[imgSize * 3];
	host_dispColorLeft = new unsigned char[imgSize * 3];
	host_dispColorRight = new unsigned char[imgSize * 3];
	host_dispRawLeft = new float[imgSize];
	host_dispRawRight = new float[imgSize];

	CudaSafeCall(hipMalloc((void**)&dev_grayLeft,imgSize));
	CudaSafeCall(hipMalloc((void**)&dev_grayRight,imgSize));
	CudaSafeCall(hipMalloc((void**)&dev_colorLeft,imgSize * 3));
	CudaSafeCall(hipMalloc((void**)&dev_colorRight,imgSize * 3));
	CudaSafeCall(hipMalloc((void**)&dev_dispColorLeft,imgSize * 3));
	CudaSafeCall(hipMalloc((void**)&dev_dispColorRight,imgSize * 3));
	CudaSafeCall(hipMalloc((void**)&dev_dispRawLeft,imgSize * sizeof(float)));
	CudaSafeCall(hipMalloc((void**)&dev_dispRawRight,imgSize * sizeof(float)));
	isInitialized = true;
	return true;
}
void cCudaStereoMatcher::deinitSystem() {
	if (isInitialized) {
		isInitialized = false;

		delete host_grayLeft;
		delete host_grayRight;
		delete host_colorLeft;
		delete host_colorRight;
		delete host_dispColorLeft;
		delete host_dispColorRight;
		delete host_dispRawLeft;
		delete host_dispRawRight;

		CudaSafeCall(hipFree((void**) &dev_colorLeft));
		CudaSafeCall(hipFree((void**) &dev_colorRight));
		CudaSafeCall(hipFree((void**) &dev_grayLeft));
		CudaSafeCall(hipFree((void**) &dev_grayRight));
		CudaSafeCall(hipFree((void**) &dev_dispColorLeft));
		CudaSafeCall(hipFree((void**) &dev_dispColorRight));
		CudaSafeCall(hipFree((void**) &dev_dispRawLeft));
		CudaSafeCall(hipFree((void**) &dev_dispRawRight));
	}
}
void cCudaStereoMatcher::updateSettings(int kernelSize, int maxDisp, int consistencyTreshold) {
	this->kernelSize = kernelSize;
	this->maxDisp = maxDisp;
	this->consistencyTreshold = consistencyTreshold;
}
bool cCudaStereoMatcher::processStereo(unsigned char* host_leftImg, unsigned char* host_rightImg) {

	if(!isInitialized)
		return false;

	bool colorInput = modes & OP_INPUT_COLOR;
	bool colorMatch = modes & OP_COLOR_MATCH;
	bool subpixel = modes & OP_SUBPIXEL;

	CudaSafeCall(hipMemset(dev_dispColorLeft,0,imgSize*3));
	CudaSafeCall(hipMemset(dev_dispColorRight,0,imgSize*3));
	CudaSafeCall(hipMemset(dev_dispRawLeft,0,imgSize*sizeof(float)));
	CudaSafeCall(hipMemset(dev_dispRawRight,0,imgSize*sizeof(float)));

	if(colorInput){
		CudaSafeCall(hipMemcpy(dev_colorLeft,host_leftImg,imgSize*3,hipMemcpyHostToDevice));
		CudaSafeCall(hipMemcpy(dev_colorRight,host_rightImg,imgSize*3,hipMemcpyHostToDevice));
		if(colorMatch && subpixel){

		}
		else if(colorMatch){
			kernelStereoMatchL2R<<<blockCnt, blockSize>>>(dev_colorLeft,
					dev_colorRight, dev_dispRawLeft, width, height, kernelSize,
					maxDisp, colorMatch, subpixel);
			CudaCheckError();
			kernelStereoMatchR2L<<<blockCnt, blockSize>>>(dev_colorLeft,
					dev_colorRight, dev_dispRawRight, width, height, kernelSize,
					maxDisp, colorMatch, subpixel);
			CudaCheckError();

			if (consistencyTreshold >= 0) {
				kernelLRConsistencyCheck<<<blockCnt, blockSize>>>(
						dev_dispRawLeft, dev_dispRawRight, width, height,
						kernelSize, consistencyTreshold, colorMatch);
				CudaCheckError();
			}
		}
		else if (subpixel){

		}else {
			kernelRGBToGray<<<blockCnt, blockSize>>>(dev_colorLeft,dev_grayLeft, imgSize);
			CudaCheckError();
			kernelRGBToGray<<<blockCnt, blockSize>>>(dev_colorRight,dev_grayRight, imgSize);
			CudaCheckError();

			kernelStereoMatchL2R<<<blockCnt, blockSize>>>(dev_grayLeft,
					dev_grayRight, dev_dispRawLeft, width, height, kernelSize,
					maxDisp, colorMatch, subpixel);
			CudaCheckError();
			kernelStereoMatchR2L<<<blockCnt, blockSize>>>(dev_grayLeft,
					dev_grayRight, dev_dispRawRight, width, height, kernelSize,
					maxDisp, colorMatch, subpixel);
			CudaCheckError();

			if (consistencyTreshold >= 0) {
				kernelLRConsistencyCheck<<<blockCnt, blockSize>>>(
						dev_dispRawLeft, dev_dispRawRight, width, height,
						kernelSize, consistencyTreshold, colorMatch);
				CudaCheckError();
			}
		}
	}
	// Grayscale
	// TODO Testen
	else{

		if (subpixel) {

		} else {
			CudaSafeCall(
					hipMemcpy(dev_grayLeft, host_leftImg, imgSize,
							hipMemcpyHostToDevice));
			CudaSafeCall(
					hipMemcpy(dev_grayRight, host_rightImg, imgSize,
							hipMemcpyHostToDevice));

			kernelStereoMatchL2R<<<blockCnt, blockSize>>>(dev_grayLeft,
					dev_grayRight, dev_dispRawLeft, width, height, kernelSize,
					maxDisp, colorMatch, subpixel);
			CudaCheckError();
			kernelStereoMatchR2L<<<blockCnt, blockSize>>>(dev_grayLeft,
					dev_grayRight, dev_dispRawRight, width, height, kernelSize,
					maxDisp, colorMatch, subpixel);
			CudaCheckError();

			if (consistencyTreshold >= 0) {
				kernelLRConsistencyCheck<<<blockCnt, blockSize>>>(
						dev_dispRawLeft, dev_dispRawRight, width, height,
						kernelSize, consistencyTreshold, colorMatch);
				CudaCheckError();
			}
		}
	}

	kernelGrayToPseudoColor<<<blockCnt,blockSize>>>(dev_dispRawLeft,dev_dispColorLeft,imgSize,maxDisp,0,120);
	CudaCheckError();
	kernelGrayToPseudoColor<<<blockCnt,blockSize>>>(dev_dispRawRight,dev_dispColorRight,imgSize,maxDisp,0,120);
	CudaCheckError();

	// Receive data
	CudaSafeCall(hipMemcpy(host_dispRawLeft,dev_dispRawLeft,imgSize * sizeof(float),hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy(host_dispRawRight,dev_dispRawRight,imgSize * sizeof(float),hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy(host_dispColorLeft,dev_dispColorLeft,imgSize*3,hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy(host_dispColorRight,dev_dispColorRight,imgSize*3,hipMemcpyDeviceToHost));

	return true;
}
