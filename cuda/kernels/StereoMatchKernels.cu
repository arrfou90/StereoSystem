#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "StereoMatchKernels.cuh"

__global__ void kernelStereoMatchL2R(unsigned char* dev_leftImg,
		unsigned char* dev_rightImg, float* dev_disparity, int width,
		int height, int kernelSize, int maxDisp) {

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int imgX = idx % width;
	int imgY = idx / width;
	int imgSize = width * height;
	int halfKS = kernelSize / 2;

	bool doCalc = imgY > halfKS && imgY < height - halfKS && imgX > halfKS
			&& imgX < width - halfKS;

	// Berechne disparitätswert für pixel idx
	if (doCalc) {
		// x,y gegeben. Berechne x2 und disparität
		int bestSAD = 65532;
		int bestDisparity = 0;
		unsigned char* pX1L = dev_leftImg + idx - halfKS - halfKS * width;// Zeiger auf Anfang des Kernel
		unsigned char* pX1R = dev_rightImg + idx - halfKS - halfKS * width;

		for (int currDisp = 0; currDisp >= -maxDisp; currDisp--) {
			if (currDisp + imgX < 0)
				break;

			int sad = 0;
			unsigned char* pL = pX1L;
			unsigned char* pR = pX1R + currDisp;

			// Berechne SAD (y,x1,x2)
			for (int dy = -halfKS; dy <= halfKS; dy++) {
				for (int dx = -halfKS; dx <= halfKS; dx++) {
					sad += abs(*pL - *pR);	// delta;

					pL++;
					pR++;
				}
				pL += width - kernelSize;
				pR += width - kernelSize;
			}

			if (sad < bestSAD) {
				bestSAD = sad;
				bestDisparity = currDisp;
			}
		}
		dev_disparity[idx] = abs(bestDisparity);
	}
	__syncthreads();

}
__global__ void kernelStereoMatchR2L(unsigned char* dev_leftImg,
		unsigned char* dev_rightImg, float* dev_disparity, int width,
		int height, int kernelSize, int maxDisp) {

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int imgX = idx % width;
	int imgY = idx / width;
	int imgSize = width * height;
	int halfKS = kernelSize / 2;

	bool doCalc = imgY > halfKS && imgY < height - halfKS && imgX > halfKS
			&& imgX < width - halfKS;

	// Berechne disparitätswert für pixel idx
	if (doCalc) {
		// x,y gegeben. Berechne x2 und disparität
		int bestSAD = 65532;
		int bestDisparity = 0;
		unsigned char* pX1L = dev_leftImg + idx - halfKS - halfKS * width;// Zeiger auf Anfang des Kernel
		unsigned char* pX1R = dev_rightImg + idx - halfKS - halfKS * width;

		for (int currDisp = 0; currDisp <= maxDisp; currDisp++) {
			if (currDisp + imgX >= width)
				break;

			int sad = 0;
			unsigned char* pL = pX1L + currDisp;
			unsigned char* pR = pX1R;

			// Berechne SAD (y,x1,x2)
			for (int dy = -halfKS; dy <= halfKS; dy++) {
				for (int dx = -halfKS; dx <= halfKS; dx++) {
					sad += abs(*pL - *pR);	// delta;

					pL++;
					pR++;
				}
				pL += width - kernelSize;
				pR += width - kernelSize;
			}

			if (sad < bestSAD) {
				bestSAD = sad;
				bestDisparity = currDisp;
			}
		}
		dev_disparity[idx] = abs(bestDisparity);
	}
	__syncthreads();

}
__global__ void kernelLRConsistencyCheck(float* dev_dispLeft,
		float* dev_dispRight, int width, int height, int kernelSize,
		int consistencyTreshold) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int imgX = idx % width;
	int imgY = idx / width;
	int imgSize = width * height;
	int halfKS = kernelSize / 2;

	bool doCalc = imgY > halfKS && imgY < height - halfKS && imgX > halfKS
			&& imgX < width - halfKS;

	if (doCalc) {
		int xR = imgX - dev_dispLeft[idx];
		if (xR < 0 || xR >= width) {
			dev_dispLeft[idx] = 0;
			dev_dispRight[idx] = 0;
		} else {
			int idxR = imgY * width + xR;
			if (abs(int(dev_dispLeft[idx] - dev_dispRight[idxR]))
					> consistencyTreshold) {
				dev_dispLeft[idx] = 0;
				dev_dispRight[idx] = 0;
			}
		}
	}
}
